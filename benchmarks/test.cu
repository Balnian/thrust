#include "hip/hip_runtime.h"
#include <string>

#include <jitify/jitify.hpp>

#define JITABLE(code) std::string program_source =  

#ifndef RUNTIME
#include <string>
std::string program_source = R"FOO( #if false
#endif

    
    template<int N, typename T>
    __global__
    void my_kernel(T* data) {
        T data0 = data[0];
        for( int i=0; i<N-1; ++i ) {
            data[0] *= data0;
        }
    };
#ifndef RUNTIME
)FOO";
#endif

int main()
{
    static jitify::JitCache kernel_cache;
    jitify::Program program = kernel_cache.program(program_source);
    int data[3] = {1,2,3};
    dim3 grid(1);
    dim3 block(1);
    using jitify::reflection::type_of;
    program.kernel("my_kernel")
        .instantiate(3, type_of(*data))
        .configure(grid, block)
        .launch(data);
}